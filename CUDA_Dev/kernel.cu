#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <chrono>
#include <stdio.h>
#include <string>
#include <string.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include <iomanip>
#include <locale.h>

using namespace std;

int main()
{
    const std::string intention = "I am love. REGULATE AND INTEGRATE. OM.";

    int num_chars = intention.length();

    thrust::host_vector<int> host_intention_vector;

    for (int i = 0; i < num_chars; ++i)
    {
        host_intention_vector.push_back(int(intention.at(i)));
    }

    long long num_iterations = 0;

    thrust::device_vector<int> device_intention_vector;

    // copy all of H back to the beginning of D
    thrust::copy(host_intention_vector.begin(), host_intention_vector.end(), device_intention_vector.begin());

    auto start = std::chrono::system_clock::now();
    auto end = std::chrono::system_clock::now();

    for (int i = 0; i < 10000; ++i) {
        for (int j = 0; j < num_chars; ++j) {
            device_intention_vector[j] = 3; //The Intention Repeater Call
        }
        end = std::chrono::system_clock::now();
        ++num_iterations;
    }
    cout << "Number times repeated: " << std::to_string(num_iterations) << endl;
    return 0;
}